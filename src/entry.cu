#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#include <cuComm/simulation.cuh>

#include "entry.h"

__global__ void test_template() {
  int idx = (blockDim.x * blockIdx.x) + threadIdx.x;

  float *test0 = nullptr;
  double *test1 = nullptr;
  thrust::complex<float> *test2 = nullptr;
  thrust::complex<double> *test3 = nullptr;

  switch (idx % 4) {
    case 0:
      simulation::awgn(test0, 0.0F);
      break;
    case 1:
      simulation::awgn(test1, 0.0);
      break;
    case 2:
      simulation::awgn(test2, 0.0F);
      break;
    case 3:
      simulation::awgn(test3, 0.0);
      break;
  }
}

int entry() {
  constexpr int threads = 256;
  constexpr int blocks = 1;

  test_template<<<blocks, threads>>>();
  hipDeviceSynchronize();
  return 0;
}
